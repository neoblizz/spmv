
#include <thrust/device_vector.h>
#include <time.h>

#include <formats/csr.hxx>
#include <iostream>
#include <string>
#include <util/display.hxx>

#include "test_cub.h"
#include "test_cusparse.h"
#include "test_moderngpu.h"
#include "test_tiled.h"
#include "test_utils.h"

enum SPMV_t { MGPU, CUB, CUSPARSE, TILED };
enum LB_t { THREAD_PER_ROW, WARP_PER_ROW, BLOCK_PER_ROW, MERGE_PATH };

template <typename index_t = int, typename value_t = float, typename hinput_t,
          typename dinput_t, typename doutput_t>
double run_test(SPMV_t spmv_impl, csr_t<index_t, value_t>& sparse_matrix,
                hinput_t& hin, dinput_t& din, doutput_t& dout,
                bool check = true, bool debug = false) {
  // Reset the output vector
  thrust::fill(dout.begin(), dout.end(), 0);

  double elapsed_time = 0;

  //   Run on appropriate GPU implementation
  if (spmv_impl == MGPU) {
    elapsed_time = spmv_mgpu(sparse_matrix, din, dout);
  } else if (spmv_impl == CUB) {
    // elapsed_time = spmv_cub(sparse_matrix, din, dout);
  } else if (spmv_impl == CUSPARSE) {
    elapsed_time = spmv_cusparse(sparse_matrix, din, dout);
  } else if (spmv_impl == TILED) {
    elapsed_time = spmv_tiled(sparse_matrix, din, dout, debug);
  } else {
    std::cout << "Unsupported SPMV implementation" << std::endl;
  }

  printf("GPU finished in %lf ms\n", elapsed_time);

  //   Copy results to CPU
  if (check) {
    thrust::host_vector<float> h_output = dout;

    // Run on CPU
    thrust::host_vector<float> cpu_ref(sparse_matrix.num_rows);
    cpu_spmv(sparse_matrix, hin, cpu_ref);

    for (index_t row = 0; row < sparse_matrix.num_rows; row++) {
      cpu_ref[row] = 0.0;
      // Loop over all the non-zeroes within A's row
      for (auto k = sparse_matrix.row_offsets[row];
           k < sparse_matrix.row_offsets[row + 1]; ++k)
        cpu_ref[row] +=
            sparse_matrix.nonzero_vals[k] * hin[sparse_matrix.col_idx[k]];
    }

    util::display(hin, "cpu_in");
    util::display(din, "gpu_in");
    util::display(cpu_ref, "cpu_out");
    util::display(dout, "gpu_out");

    // Validate
    bool passed = validate(h_output, cpu_ref);
    if (passed) {
      std::cout << "Validation Successful" << std::endl;
      return elapsed_time;
    } else {
      std::cout << "Validation Failed" << std::endl;
      return -1;
    }
  }
  return elapsed_time;
}

int main(int argc, char** argv) {
  /* ========== PREPARE DATA ========== */
  bool debug = false;

  // Read in matrix market file
  std::string filename = argv[1];

  // Construct a csr matrix from the mtx file
  csr_t<int, float> sparse_matrix;

  std::cout << "Loading from Matrix Market File" << std::endl;
  std::cout << filename << std::endl;
  sparse_matrix.build(filename);

  util::display(sparse_matrix, "sparse_matrix");

  thrust::host_vector<float> h_input(sparse_matrix.num_columns);
  srand(0);
  srand(time(NULL));
  for (size_t v = 0; v < h_input.size(); v++) h_input[v] = rand() % 64;

  thrust::device_vector<float> d_input = h_input;  // Only needs to occur once
  thrust::device_vector<float> d_output(sparse_matrix.num_rows);

  std::cout << std::endl << std::endl;

  /* ========== RUN SPMV ========== */

  // GPU SPMV
  // std::cout << "Running ModernGPU" << std::endl;
  // double elapsed_mgpu =
  //     run_test(MGPU, sparse_matrix, h_input, d_input, d_output);

  std::cout << "===== Running cuSparse =====" << std::endl;
  double elapsed_cusparse =
      run_test(CUSPARSE, sparse_matrix, h_input, d_input, d_output, true, debug);

  std::cout << std::endl << std::endl;

  // NOTE: CUB appears to have a bug at the moment. I have filed an issue
  // on the github repository
  // std::cout << "Running CUB" << std::endl;
  // double elapsed_cub = run_test(CUB, sparse_matrix, h_input, d_input,
  // d_output);

//  std::cout << "===== Running Tiled =====" << std::endl;
//  double elapsed_tiled =
//      run_test(TILED, sparse_matrix, h_input, d_input, d_output, true,debug);

  printf("%s,%d,%d,%d,%f\n", filename.c_str(), sparse_matrix.num_rows,
         sparse_matrix.num_columns, sparse_matrix.num_nonzeros,
         elapsed_cusparse);

  return 0;
}
